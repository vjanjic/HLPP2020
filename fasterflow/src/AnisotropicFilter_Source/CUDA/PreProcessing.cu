#include "hip/hip_runtime.h"
/*
*
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>


// includes, project
//#include <cutil_inline.h>

// includes, kernels
//#include "Anisotropicfilter_kernel.cu"
__global__ void anisotropy_kernel(float1* imInD, int M,int N, float k, float lambda, short type);

#define BLOCK_SIZE 16

int N,M;
int iter;
double k,lambda;
float1* imInD;
int size;
short otype;





extern "C" void
initPreProcessing(int imWidth, int imHeight, int niter, float kappa, float lampda, int type)
{

	N = imWidth;
    M = imHeight;
	
	iter = niter;  
    k = kappa;  
    lambda = lampda;   
    otype = type;  

	
	hipMalloc((void **)&imInD,M*N*sizeof(float));


}


extern "C" void
executePreProcessing(float* imIn, float* output, int len)
{

	/* execute anisotropie filter */
	hipMemcpy(imInD, imIn,len*sizeof(float), hipMemcpyHostToDevice);
    
    dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
    dim3 dimGrid(ceil(N / (float)dimBlock.x), ceil(M / (float)dimBlock.y));    

    for (int i=1;i<=iter;++i) {
        anisotropy_kernel<<<dimGrid,dimBlock>>>(imInD, M, N,(float)k,(float)lambda, otype);
    }

 hipDeviceSynchronize();
//hipError_t cerr = hipGetLastError();
//hipGetErrorString(cerr);
	//CUT_CHECK_ERROR("Kernel execution failed");
	
   
    hipMemcpy(output,imInD,len*sizeof(float),hipMemcpyDeviceToHost);  
	
	hipDeviceReset();
	
	//cutilSafeCall( hipFree( d_Temp ) );    
	
}



  
