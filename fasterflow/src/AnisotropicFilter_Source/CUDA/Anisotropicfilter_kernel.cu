
#include <hip/hip_runtime.h>


__global__ void anisotropy_kernel(float1* imInD, int M,int N, float k, float lambda, short type) {
         
	     int i = blockIdx.x * blockDim.x + threadIdx.x;
         int j = blockIdx.y * blockDim.y + threadIdx.y;
         int index = j+i*M;
         int len = N*M;

        float deltaN; 
		float deltaS;
		float deltaW;
		float deltaE;
		float cN; 
		float cS;
		float cW;
		float cE;
		int indexN; 
		int indexS;
		int indexW;
		int indexE;
		float val;
    
        
        indexN = (j)+(i-1)*(M);
        indexS = (j)+(i+1)*(M);
        indexW = (j-1)+(i)*(M);
        indexE = (j+1)+(i)*(M);        

        if (i>1)
            deltaN = imInD[indexN].x-imInD[index].x;
            
        if (indexS < len)
            deltaS = imInD[indexS].x-imInD[index].x;  
              
        if (j>1)
            deltaW = imInD[indexW].x-imInD[index].x;   
             
        if (indexE < len)
            deltaE = imInD[indexE].x-imInD[index].x;   
      
        if (type==1) {
            cN = exp(-(pow((deltaN / k),2)));
            cS = exp(-(pow((deltaS / k),2)));
            cW = exp(-(pow((deltaW / k),2)));
            cE = exp(-(pow((deltaE / k),2)));  
        } else {
            cN = 1/(1+pow((deltaN / k),2));
            cS = 1/(1+pow((deltaS / k),2));
            cW = 1/(1+pow((deltaW / k),2));
            cE = 1/(1+pow((deltaE / k),2));        
        }

		
		val = (lambda*(cN*deltaN + cS*deltaS + cW*deltaW + cE*deltaE));  
		imInD[index].x += val;

        
		
        __syncthreads();    
}




